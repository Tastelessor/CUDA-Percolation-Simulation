#include "hip/hip_runtime.h"
// -*- mode: C++; -*-
//
// Copyright (C) 2022, Rupert Nash, The University of Edinburgh.
//
// All rights reserved.
//
// This file is provided to you to complete an assessment and for
// subsequent private study. It may not be shared and, in particular,
// may not be posted on the internet. Sharing this or any modified
// version may constitute academic misconduct under the University's
// regulations.

#include "perc_gpu.h"

#include <cstdio>
#include <cstring>
#include <vector>

constexpr int printfreq = 100;

// Do the 2D indexing into the array.
//
// Assumes that you have a variable `N` in scope specifying the the
// size of the non-halo part of the grid.
#define get(array, i, j) array[(i)*(N+2) + j]
#define INT sizeof(int)

__global__ void update_by_neighbours(int* current_state, int* next_state, int N, int *dchange) {
    int row, col;

    row = blockIdx.y * blockDim.y + threadIdx.y + 1;
    col = blockIdx.x * blockDim.x + threadIdx.x + 1;

    int old_val = current_state[(row)*(N+2)+col];
    int new_val = old_val;

    if (old_val != 0) {
        new_val = current_state[(row-1)*(N+2)+col] > new_val ? current_state[(row-1)*(N+2)+col] : new_val;
        new_val = current_state[(row+1)*(N+2)+col] > new_val ? current_state[(row+1)*(N+2)+col] : new_val;
        new_val = current_state[(row)*(N+2)+col-1] > new_val ? current_state[(row)*(N+2)+col-1] : new_val;
        new_val = current_state[(row)*(N+2)+col+1] > new_val ? current_state[(row)*(N+2)+col+1] : new_val;
    }

    if (new_val != old_val) {
        *dchange = atomicAdd(dchange, 1);
        next_state[row * (N + 2) + col] = new_val;
    }
}

// Perform a single step of the algorithm.
//
// For each point (if fluid), set it to the maximum of itself and the
// four von Neumann neighbours.
//
// Returns the total number of changed cells.
int percolate_gpu_step(int M, int N, int const *state, int *next) {
    int nchange = 0;

    for (int i = 1; i <= M; ++i) {
        for (int j = 1; j <= N; ++j) {
            int const oldval = get(state, i, j);
            int newval = oldval;

            // 0 => solid, so do nothing
            if (oldval != 0) {
                // Set next[i][j] to be the maximum value of state[i][j] and
                // its four nearest neighbours
                newval = std::max(newval, get(state, i - 1, j));
                newval = std::max(newval, get(state, i + 1, j));
                newval = std::max(newval, get(state, i, j - 1));
                newval = std::max(newval, get(state, i, j + 1));

                if (newval != oldval) {
                    ++nchange;
                }
            }

            next[(i) * (N + 2) + j] = newval;
        }
    }
    return nchange;
}

// Given an array, state, of size (M+2) x (N+2) with a halo of zeros,
// iteratively perform percolation of the non-zero elements until no
// changes or 4 *max(M, N) iterations.
void percolate_gpu(int M, int N, int *state) {
    int const npoints = (M + 2) * (N + 2);
    // Temporary work array
    std::vector<int> temp(npoints);
    // Copy the initial state to the temp, only the halos are
    // *required*, but much easier this way!
    std::memcpy(temp.data(), state, INT * npoints);

    int const maxstep = 4 * std::max(M, N);
    int step = 1;
    int nchange = 1;

    // Use pointers to the buffers (which we swap below) to avoid copies.
//    int *current = state;
//    int *next = temp.data();
    int *current;
    int *next;
    int *dchange;

    // Print device details
    int deviceNum;
    hipGetDevice(&deviceNum);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceNum);
    std::printf("Device name: %s\n", prop.name);

    // Allocate memory on device
    size_t memory_size = npoints * INT;
    hipMalloc(&current, memory_size);
    hipMalloc(&next, memory_size);
    hipMalloc(&dchange, INT);
    hipMemcpy(current, state, memory_size, hipMemcpyHostToDevice);
    hipMemcpy(next, temp.data(), memory_size, hipMemcpyHostToDevice);
    hipMemcpy(dchange, &nchange, INT, hipMemcpyHostToDevice);

    // GPU decomposition
    const dim3 threadsPerBlock(256, 256, 1);
    const dim3 blocksPerGrid(N/256, M/256, 1);

    while (nchange && step <= maxstep) {
        nchange = 0;
        update_by_neighbours<<<blocksPerGrid, threadsPerBlock>>> (current, next, N, dchange);
        hipDeviceSynchronize();
//        nchange = percolate_gpu_step(M, N, current, next);
        hipMemcpy(&nchange, dchange, INT, hipMemcpyDeviceToHost);

        //  Report progress every now and then
        if (step % printfreq == 0) {
            std::printf("percolate: number of changes on step %d is %d\n",
                        step, nchange);
        }

        // Swap the pointers for the next iteration
//        std::swap(next, current);
        hipMemcpy(current, next, memory_size, hipMemcpyDeviceToHost);
        step++;
    }

    hipMemcpy(state, current, memory_size, hipMemcpyDeviceToHost);

    // Answer now in `current`, if that's not the same pointer as
    // `state`, have to copy out.
/*    if (current != state) {
        std::memcpy(state, temp.data(), INT * npoints);
    }*/
    std::memcpy(state, temp.data(), INT * npoints);

    hipFree(current);
    hipFree(next);
    hipFree(dchange);
}
